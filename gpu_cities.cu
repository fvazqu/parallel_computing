#include "hip/hip_runtime.h"
#include <iostream>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif
#include <fstream>
#include <sstream>
#include <cmath>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA kernel to calculate distance using the Haversine formula
__global__
void haversine_kernel(double* latitudes, double* longitudes, double start_lat, double start_lon, double* distances, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        double r = 3958.8;  // Radius of Earth in miles

        double lat1_r = start_lat * M_PI / 180.0;
        double long1_r = start_lon * M_PI / 180.0;
        double lat2_r = latitudes[tid] * M_PI / 180.0;
        double long2_r = longitudes[tid] * M_PI / 180.0;

        double a = pow(sin((lat2_r - lat1_r) / 2), 2) + cos(lat1_r) * cos(lat2_r) * pow(sin((long2_r - long1_r) / 2), 2);
        double c = 2 * atan2(sqrt(a), sqrt(1 - a));
        distances[tid] = r * c;
    }
}


// CUDA kernel to calculate distance using the Haversine formula
__global__
void calculate_distances_from_cairo(double* latitudes, double* longitudes, double cairo_lat, double cairo_lon, double* distances, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        double r = 3958.8;  // Radius of Earth in miles

        double lat1_r = cairo_lat * M_PI / 180.0;
        double long1_r = cairo_lon * M_PI / 180.0;
        double lat2_r = latitudes[tid] * M_PI / 180.0;
        double long2_r = longitudes[tid] * M_PI / 180.0;

        double a = pow(sin((lat2_r - lat1_r) / 2), 2) + cos(lat1_r) * cos(lat2_r) * pow(sin((long2_r - long1_r) / 2), 2);
        double c = 2 * atan2(sqrt(a), sqrt(1 - a));
        distances[tid] = r * c;
    }
}

// Define the City Struct
struct City {
    std::string name;
    double latitude;
    double longitude;
};

// Initiazlize the cities vector
std::vector<City> cities;


int main(int argc, char* argv[]) {
    // Set up configurations
    int start_row = 1;
    int end_row = 47868;
    int columns[] = { 0, 2, 3 };
    double start_lat = 26.3017;
    double start_lon = -98.1633;

    // Read CSV file and populate arrays
    std::string filePath = "C:\\Users\\fvazq\\OneDrive\\Documents\\c++projects\\worldcities.csv";
    std::ifstream file(filePath);
    std::vector<std::string> names;
    std::vector<double> latitudes;
    std::vector<double> longitudes;


    // Part 1
    if (!file.is_open()) {
        std::cerr << "Failed to open file at " << filePath << std::endl;
        return -1;
    }

    if (file.is_open()) {
        std::string line;
        int current_row = 0;
        while (std::getline(file, line)) {
            if (current_row >= start_row && current_row <= end_row) {
                std::istringstream iss(line);
                std::string token;
                City city;
                int col_index = 0;
                while (std::getline(iss, token, ',') && col_index <= 3) {
                    if (col_index == columns[0]) {
                        city.name = token;
                        names.push_back(token);
                    }
                    else if (col_index == columns[1]) {
                        city.latitude = std::stod(token);
                        latitudes.push_back(std::stod(token));
                    }
                    else if (col_index == columns[2]) {
                        city.longitude = std::stod(token);
                        longitudes.push_back(std::stod(token));
                    }
                    col_index++;
                }
                cities.push_back(city);
                if (current_row == end_row) {
                    break;
                }
            }
            current_row++;
        }
        file.close();
    }

    // Allocate memory on the GPU
    double* latitudes_gpu;
    double* longitudes_gpu;
    double* distances_gpu;
    hipMalloc(&latitudes_gpu, end_row * sizeof(double));
    hipMalloc(&longitudes_gpu, end_row * sizeof(double));
    hipMalloc(&distances_gpu, end_row * sizeof(double));

    // Copy data from host to device
    hipMemcpy(latitudes_gpu, latitudes.data(), end_row * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(longitudes_gpu, longitudes.data(), end_row * sizeof(double), hipMemcpyHostToDevice);

    // Measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Calculate grid and block sizes
    int blockSize = 256;
    int numBlocks = (end_row + blockSize - 1) / blockSize;

    // Launch the kernel
    haversine_kernel << <numBlocks, blockSize >> > (latitudes_gpu, longitudes_gpu, start_lat, start_lon, distances_gpu, end_row);

    // Copy results from device to host
    std::vector<double> distances(end_row);
    hipMemcpy(distances.data(), distances_gpu, end_row * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(latitudes_gpu);
    hipFree(longitudes_gpu);
    hipFree(distances_gpu);


    // Part 2
    // Count cities within 1000 miles
    int count = 0;
    for (int i = 0; i < end_row; i++) {
        if (distances[i] <= 1000 && distances[i] > 0) {
            count += 1;
        }
    }

    std::cout << "Total count of cities within 1000 miles of Edinburg, TX: " << count << std::endl;


    // Part 3
    // Find Cairo Egypt Coordinates
    double st_lat = 30.0444;
    double st_lon = 31.2358;

    // Allocate memory for distances array on GPU
    double* distances2_gpu;
    hipMalloc(&distances2_gpu, end_row * sizeof(double));

    // Copy Cairo's latitude and longitude to the GPU
    double cairo_lat_gpu = st_lat;
    double cairo_lon_gpu = st_lon;

    // Calculate grid and block sizes
    //int blockSize = 256;
    //int numBlocks = (end_row + blockSize - 1) / blockSize;

    // Launch the kernel to calculate distances from Cairo
    calculate_distances_from_cairo << <numBlocks, blockSize >> > (latitudes_gpu, longitudes_gpu, cairo_lat_gpu, cairo_lon_gpu, distances2_gpu, end_row);

    // Copy the distances array back to the host
    std::vector<double> distances2(end_row);
    hipMemcpy(distances2.data(), distances2_gpu, end_row * sizeof(double), hipMemcpyDeviceToHost);

    // Find the closest city to Cairo
    double min_distance = 100000;
    int closest_index = 0;
    for (int i = 0; i < end_row; ++i) {
        if (distances2[i] < min_distance && distances2[i] > 0) {
            min_distance = distances2[i];
            closest_index = i;
        }
    }

    // Output the closest city to Cairo
    std::cout << "Closest City to Cairo, Egypt: " << cities[closest_index].name << ", Distance: " << min_distance << " miles" << std::endl;

    // Free memory on the GPU
    hipFree(distances2_gpu);



    // Measure execution time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;

    // Print execution time
    std::cout << "Execution time: " << duration.count() << " milliseconds" << std::endl;

    return 0;
}
